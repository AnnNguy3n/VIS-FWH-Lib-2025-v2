#include "hip/hip_runtime.h"
#pragma once
#include "../../Generator/HomoPoly/CUDA/HomoPolyMethod.cu"


#ifndef _NUM_THRESHOLD_PER_CYCLE_
#define _NUM_THRESHOLD_PER_CYCLE_
const int __NUM_THRESHOLD_PER_CYCLE__ = 10;
#endif


__device__ __forceinline__ double max_of_array(double *array, int left, int right, double supremum){
    double max_ = __NEGATIVE_INFINITY__;
    for (int i=left; i<right; i++){
        if (array[i] < supremum && array[i] > max_) max_ = array[i];
    }
    return max_;
}


__device__ __forceinline__ void top_n_of_array(double *array, int left, int right, double *result, int start, int n){
    double supremum = __POSITIVE_INFINITY__;
    for (int i=0; i<n; i++){
        supremum = max_of_array(array, left, right, supremum);
        result[start+i] = supremum;
    }
}


__global__ void fill_thresholds(double *weights, double *thresholds, int *INDEX, int index_length, int num_array, int length){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_cycle = index_length - 2;
    if (index < num_array*num_cycle){
        int ix = index % num_cycle;
        int iy = index / num_cycle;
        top_n_of_array(weights + iy*length,
                       INDEX[ix+1], INDEX[ix+2],
                       thresholds + iy*__NUM_THRESHOLD_PER_CYCLE__*num_cycle,
                       ix*__NUM_THRESHOLD_PER_CYCLE__, __NUM_THRESHOLD_PER_CYCLE__);
    }
}


__device__ __forceinline__ int binary_symbol_search(int *SYMBOL, int start, int end, int target){
    int left = start, right = end-1;
    int mid;
    while (left <= right){
        mid = left + (right - left) / 2;
        if (SYMBOL[mid] == target) return mid;
        if (SYMBOL[mid] < target) left = mid + 1;
        else right = mid - 1;
    }
    return -1;
}


__device__ __forceinline__ void _double_year_threshold_investing(double *weight, double threshold, int t_idx, double *result,
    double INTEREST, int *INDEX, double *PROFIT, int *SYMBOL, int *BOOL_ARG, int index_size, int num_cycle){
    int reason;
    double Geo2 = 0, Har2 = 0;
    int start, end, end2, count, k, sym, s, rs_idx;
    double temp, n;
    bool check;

    check = false;
    start = INDEX[index_size - 2];
    end = INDEX[index_size - 1];
    for (k=start; k<end; k++){
        if (weight[k] > threshold){
            check = true;
            break;
        }
    }
    reason = !check;

    for (int i=index_size-3; i>0; i--){
        start = INDEX[i];
        end = INDEX[i+1];
        end2 = INDEX[i+2];
        temp = 0;
        count = 0;
        check = false;

        for (k=start; k<end; k++){
            if (weight[k] > threshold){
                check = true;
                if (!BOOL_ARG[k]) continue;

                if (reason){
                    count++;
                    temp += PROFIT[k];
                }
                else {
                    sym = SYMBOL[k];
                    s = binary_symbol_search(SYMBOL, end, end2, sym);
                    if (s != -1 && weight[s] > threshold){
                        count++;
                        temp += PROFIT[k];
                    }
                }
            }
        }

        if (!count){
            Geo2 += log(INTEREST);
            Har2 += 1.0 / INTEREST;
        } else {
            temp /= count;
            Geo2 += log(temp);
            Har2 += 1.0 / temp;
        }

        reason = !check;

        if (i <= num_cycle && t_idx+1 >= i){
            rs_idx = num_cycle - i;
            n = index_size - 2 - i;
            result[2*rs_idx] = exp(Geo2/n);
            result[2*rs_idx+1] = n / Har2;
        }
    }
}


__global__ void double_year_threshold_investing(double *weights, double *thresholds, double *results, int num_array, int num_threshold,
    int length, int num_cycle, double INTEREST, int *INDEX, double *PROFIT, int *SYMBOL, int *BOOL_ARG, int index_size){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_array*num_threshold){
        int ix = index % num_threshold;
        int iy = index / num_threshold;
        _double_year_threshold_investing(
            weights + iy*length,
            thresholds[iy*num_threshold + ix],
            ix / __NUM_THRESHOLD_PER_CYCLE__,
            results + iy*num_threshold*num_cycle*2 + ix*num_cycle*2,
            INTEREST, INDEX, PROFIT, SYMBOL, BOOL_ARG, index_size, num_cycle
        );
    }
}


__global__ void find_best_results(double *results, double *thresholds, double *finals, int num_array, int num_threshold, int num_cycle){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < 2*num_array*num_cycle){
        int iz = index % 2;
        int ix = (index/2) % num_cycle;
        int iy = (index/2) / num_cycle;

        double *result = results + iy*num_threshold*num_cycle*2;
        double *threshold = thresholds + iy*num_threshold;
        double *final_ = finals + iy*num_cycle*4 + ix*4;

        final_[2*iz] = threshold[0];
        final_[2*iz + 1] = result[2*ix + iz];
        for (int i=1; i<num_threshold; i++){
            if (result[i*num_cycle*2 + 2*ix + iz] > final_[2*iz + 1]){
                final_[2*iz] = threshold[i];
                final_[2*iz + 1] = result[i*num_cycle*2 + 2*ix + iz];
            }
        }
    }
}
